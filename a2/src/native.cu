
#include <hip/hip_runtime.h>
#include <chrono>
#include <cmath>
#include <iomanip>
#include <iostream>
#include <stdio.h>
#include <stdlib.h>
#include <fstream>
#include <vector>
#include <cmath>

/*
** Features to implement
* 1. Implement cuBLAS.
* 2. Implement A^T * x for M = N.
* 3. Implement A * B natively and using cuBLAS.
* 4. Write the report. The results were recorded earlier.
*/

const int block_size = 128;
//#define DEBUG
//#define DEBUG_SIZE 4
#define VERBOSE

/*
 * Execution example: ./task1 -min 100 -max 10000 -repeat 20
 */

/*
 * @brief	computing matrix-vector product.
 * @param       M       the number of columns,
 * @param       N       the number of rows,
 * @param       A       the matrix multiplying with,
 * @param	x	the vector multiplying with,
 * @param	b	the resulting vector. 
 */
__global__ void compute_triad(const int    M,
                              const int    N,
                              const float *A,
                              const float *x,
                              float *      b)
{
  const unsigned int i = threadIdx.x + blockIdx.x * blockDim.x;
  float sum = 0;
  if (i < M) {
    for (unsigned int j = 0; j < N; ++j) {
      sum += A[i + j * M] * x[j];
#if defined(DEBUG) && defined(VERBOSE)
      printf("A[%d] * x[%d] = b[%d]: %f * %f = %f\n", \
              i + j * M, j, i, A[i + j * M], x[j], sum);
#endif
    }
    b[i] = sum;
    sum = 0;
  }
}





/*
 * @brief      setting elements in a vector
 * @param       N       the number of elements,
 * @param       val     the value to set everywhere,
 * @param       x       the vector to set values in.
 */
__global__ void set_vector(const int N, const float val, float *x)
{
  const unsigned int idx = threadIdx.x + blockIdx.x * blockDim.x;
  if (idx < N) {
    if (val == 0) {
      x[idx] = val;
    } else { 
#ifdef DEBUG // inc vals
      x[idx] = idx + 1;
#else
      x[idx] = val;
#endif
    }
  }
}





/*
 * @brief      setting elements row-major-wise in a matrix
 * @param       M       the number of columns,
 * @param       N       the number of rows,
 * @param       val     the value to set everywhere,
 * @param       x       the matrix to set values in.
 */
__global__ void set_matrix_rowmaj(const int M, const int N, const float val, float *x)
{
  const unsigned int i = threadIdx.x + blockIdx.x * blockDim.x;
  if (i < M)
    for (unsigned int j = 0; j < N; j++) {
#ifdef DEBUG // inc vals
      x[i * N + j] = i * N + j + 1;
#else
      x[i * N + j] = val;
#endif
    }
  }




  
/*
 * @brief      setting elements column-major-wise in a matrix
 * @param       M       the number of columns,
 * @param       N       the number of rows,
 * @param       val     the value to set everywhere,
 * @param       x       the matrix to set values in.
 */
__global__ void set_matrix_colmaj(const int M, const int N, const float val, float *x)
{
  const unsigned int i = threadIdx.x + blockIdx.x * blockDim.x;
  if (i < N)
    for (unsigned int j = 0; j < M; j++) {
#ifdef DEBUG // inc vals
      x[i * M + j] = i + j * N + 1;
#else
      x[i * M + j] = val;
#endif
  }
}





/*
 * @brief       printing a matrix (in row-major format)
 * @param       M       the number of columns,
 * @param       N       the number of rows,
 * @param       x       the matrix to print.
 */
void print_matrix(const int M, const int N, std::vector<float> x)
{
  printf("\n");
  for (int i = 0; i < M; i++) {
    for (int j = 0; j < N; j++)
      printf("%f ", x.at(i + j * M));
    printf("\n");
  }
  printf("\n");
}





/*
 * @brief	printing a vector
 * @param	N	the vector length,
 * @param	x	the vector to print.
 */
void print_vector(const int N, std::vector<float> x)
{
  printf("[");
  for (int i = 0; i < N - 1; i++) {
    printf("%f, ", x.at(i));
  }
  printf("%f]\n\n", x.at(N - 1));
}





/*
 * @brief 	performs matrix and vector multiplications on the GPU
		according to the specified settings.		
 * @param	M	the number of rows in the matrix,
 * @param	N	the number of columns in the matrix,
 * @param	repeat	repetitions used to minimize noise.	
 */
float benchmark_triad(const std::size_t M,
                     const std::size_t N,
                     const long long   repeat)
{
  float *A, *x, *b;

  // allocate memory on the device
  hipMalloc(&A, M * N * sizeof(float));
  hipMalloc(&x, N * sizeof(float));
  hipMalloc(&b, M * sizeof(float));
  
  unsigned int n_blocks = (M * N + block_size - 1) / block_size;
  
  set_matrix_colmaj<<<n_blocks, block_size>>>(M, N, 1.f, A);
  n_blocks = (N + block_size - 1) / block_size;
  set_vector<<<n_blocks, block_size>>>(N, 1.f, x);
  n_blocks = (M + block_size - 1) / block_size;
  set_vector<<<n_blocks, block_size>>>(M, 0.f, b);
  
  std::vector<float> result_host(M);
  std::vector<float> A_host(M * N);
  std::vector<float> x_host(N);

#ifdef DEBUG
  const unsigned int           n_tests = 1;
  const unsigned long long int n_repeat = 1;
#else
  const unsigned int           n_tests = 20;
  const unsigned long long int n_repeat = 20; // replace 20 with repeat
#endif

  float best = 1e10, worst = 0, avg = 0;
  
  for (unsigned int t = 0; t < n_tests; ++t)
    {
      // type of t1: std::chrono::steady_clock::time_point
      const auto t1 = std::chrono::steady_clock::now();
      
      set_vector<<<n_blocks, block_size>>>(M, 0.f, b);

      for (unsigned int rep = 0; rep < n_repeat; ++rep)
        compute_triad<<<n_blocks, block_size>>>(M, N, A, x, b);

      hipDeviceSynchronize();
      // measure the time by taking the difference between the time point
      // before starting and now
      const float time =
        std::chrono::duration_cast<std::chrono::duration<float>>(
          std::chrono::steady_clock::now() - t1)
          .count();
      
      best  = std::min(best, time / n_repeat);
      worst = std::max(worst, time / n_repeat);
      avg += time / n_repeat;
    }
    
  // copy the result back to the host
  hipMemcpy(result_host.data(), b, M * sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy(A_host.data(), A, M * N * sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy(x_host.data(), x, N * sizeof(float), hipMemcpyDeviceToHost);

#ifdef DEBUG
  printf("\nA =");
  print_matrix(M, N, A_host);
  printf("x = ");
  print_vector(N, x_host);
  printf("b = ");
  print_vector(M, result_host);

  float res = 180;
  if ((abs(result_host[0] + result_host[M - 1] - res)) > 0.00000001)
    std::cout << "Error in computation, got "
              << (result_host[0] + result_host[M - 1]) << " instead of "
              << res
              << std::endl;
  else
    std::cout << "*** Congratulations! You know basic linear algebra! ***\n" << std::endl;
#else
  if (abs((result_host[0] + result_host[M - 1] - 2 * N)) > 0.00000001)
    std::cout << "Error in computation, got "
              << (result_host[0] + result_host[M - 1]) << " instead of "
              << 2 * N
              << std::endl;
#endif

  // free the memory on the device
  hipFree(A);
  hipFree(x);
  hipFree(b);

#ifdef VERBOSE
  std::cout << "(M, N) = (" << M << ", " << N << ")"
            << std::setw(8) << " - min/avg/max: "
            << std::setw(11) << best << " / "
            << std::setw(11) << avg / n_tests << " / "
            << std::setw(11) << worst
            << " seconds, or " << std::setw(8) << M * 1e-6 / best
            << " MUPD/s, or " << std::setw(8) << (M * N + M + N) * sizeof(float) * 1e-9 / best
            << " GB/s, or " << std::setw(8) << M * N * 2 / best
            << " FLOP/s" << std::endl;
#endif

/*
 ** LEGEND FOR THE METRICS USED

 ** MUPDS/s:
 * M * 1e-6 for storing the results.

 ** GB/s:
 * M * N reads from the matrix,
 * N reads from the vector, and
 * M writes for the result.

 ** FLOP/s:
 * N multiplications and additions to get one element,
 * M such operations to get one column, and
 * K such operations to get row,
 * (for the matrix-vector multiplication K = 1)
 * divided by the shortest computation time.
*/

  // returning the GB/s to write to the csv file
  return (M * N + M + N) * sizeof(float) * 1e-9 / best; 
}





/*
 * @brief	Calling the kernel with the settings in argv.
 * @param	argc	The number of arguments used when calling main.
 * @param	argv	The argument values used. These are follows
 *		
 *	min	lower limit of N,
 *	max	upper limit of N,
 * 	repeat	repetitions used to minimize noise.
 */
int main(int argc, char **argv)
{
  if (argc % 2 == 0)
    {
      std::cout << "Error, expected odd number of common line arguments"
                << std::endl
                << "Expected line of the form" << std::endl
                << "-min 100 -max 10000 -repeat 20" << std::endl;
      std::abort();
    }

  long N_min  = 8;
  long N_max  = -1;
  long repeat = -1;
  long m, n;

  // parse from the command line
  for (unsigned int l = 1; l < argc; l += 2)
    {
      std::string option = argv[l];
      if (option == "-min")
        N_min = static_cast<long>(std::stod(argv[l + 1]));
      else if (option == "-max")
        N_max = static_cast<long>(std::stod(argv[l + 1]));
      else if (option == "-repeat")
        repeat = std::atoll(argv[l + 1]);
      else
        std::cout << "Unknown option " << option << " - ignored!" << std::endl;
    }

  if (N_min < 1)
    {
      std::cout << "Expected positive size for min argument, got " << N_min
                << std::endl;
      return 0;
    }

  if (N_max < N_min)
    N_max = N_min;

#ifdef DEBUG
  m = DEBUG_SIZE;
  n = m;
  benchmark_triad(m, n, repeat);
#else
  int part = 2;
  std::ofstream myfile;

  if (part == 0)
  {
  /* saving the csvs in a seprate folder,
     please create it if you don't have it */
  myfile.open("/home/oskart/abp/a2/csv/native_square.csv");
    for (n = N_min; n <= N_max; n = (1 + n * 1.1))
    {   
        // round up to nearest multiple of 8
        n = (n + 7) / 8 * 8;
        m = n;
        myfile << n;
        myfile << " ";
        myfile << benchmark_triad(m, n, repeat);
        myfile << std::endl;
    }   
  }
  else if (part == 1)
  {
  myfile.open("/home/oskart/abp/a2/csv/native_rect1.csv");
    n = 10000;
    for (m = N_min; m <= N_max; m = (1 + m * 1.1))
    {   
        // round up to nearest multiple of 8
        m = (m + 7) / 8 * 8;
        myfile << m;
        myfile << " ";
        myfile << benchmark_triad(m, n, repeat);
        myfile << std::endl;
    }   
  }
  else if (part == 2)  
  {
    myfile.open("/home/oskart/abp/a2/csv/native_rect2.csv");
    m = 16384;
    for (n = N_min; n <= N_max; n = (1 + n * 1.1))
    {   
        // round up to nearest multiple of 8
        n = (n + 7) / 8 * 8;
        myfile << n;
        myfile << " ";
        myfile << benchmark_triad(m, n, repeat);
        myfile << std::endl;
    }   
  }
  myfile.close();
#endif

  return 0;
}
